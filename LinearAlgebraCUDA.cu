#include "hip/hip_runtime.h"
#include "linearAlgebraCUDA.cuh"

namespace WhydahGally
{
	namespace Maths
	{
		//Kernels.
		__global__ void MultipMatrices(float* d_inFirst, float* d_inSecond, float* d_out, int rowFirst, int colFirst, int colSecond)
		{
			int row = blockIdx.y * blockDim.y + threadIdx.y;
			int col = blockIdx.x * blockDim.x + threadIdx.x;

			float sum = 0.0f;

			if (row >= rowFirst || col >= colSecond)
			{
				return;
			}

			for (int i = 0; i < colFirst; i++)
			{
				sum += d_inFirst[row * colFirst + i] * d_inSecond[i * colSecond + col];
			}

			d_out[row * colSecond + col] = sum;
		}

		__global__ void MultipMatricesSH(float* d_inFirst, float* d_inSecond, float* d_out, int rowFirst, int colFirst, int colSecond)
		{
			int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
			int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

			__shared__ float shFirst[BLOCK_SIZE * BLOCK_SIZE];
			__shared__ float shSecond[BLOCK_SIZE * BLOCK_SIZE];

			float sum = 0.0f;

			for (int i = 0; i < (BLOCK_SIZE + colFirst - 1) / BLOCK_SIZE; i++)
			{
				if (i * BLOCK_SIZE + threadIdx.x < colFirst && row < rowFirst)
				{
					shFirst[threadIdx.y * BLOCK_SIZE + threadIdx.x] = d_inFirst[row * colFirst + i * BLOCK_SIZE + threadIdx.x];
				}
				else
				{
					shFirst[threadIdx.y * BLOCK_SIZE + threadIdx.x] = 0.0f;
				}

				if (i * BLOCK_SIZE + threadIdx.y < colFirst && col < colSecond)
				{
					shSecond[threadIdx.y * BLOCK_SIZE + threadIdx.x] = d_inSecond[(i * BLOCK_SIZE + threadIdx.y) * colSecond + col];
				}
				else
				{
					shSecond[threadIdx.y * BLOCK_SIZE + threadIdx.x] = 0.0f;
				}

				__syncthreads();

				for (int j = 0; j < BLOCK_SIZE; j++)
				{
					sum += shFirst[threadIdx.y * BLOCK_SIZE + j] * shSecond[j * BLOCK_SIZE + threadIdx.x];
				}

				__syncthreads();
			}

			if (row < rowFirst && col < colSecond)
			{
				d_out[((blockIdx.y * blockDim.y + threadIdx.y) * colSecond) + (blockIdx.x * blockDim.x) + threadIdx.x] = sum;
			}
		}

		__global__ void Transp(float* d_inMatrix, float* d_out, int rowMatrix, int colMatrix)
		{
			int row = blockIdx.y * blockDim.y + threadIdx.y;
			int col = blockIdx.x * blockDim.x + threadIdx.x;

			if (row >= rowMatrix || col >= colMatrix)
			{
				return;
			}

			d_out[col * rowMatrix + row] = d_inMatrix[row * colMatrix + col];
		}

		__global__ void TranspSH(float *d_inMatrix, float *d_out, int rowMatrix, int colMatrix)
		{
			int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
			int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

			__shared__ float shTile[BLOCK_SIZE * BLOCK_SIZE];

			if (blockIdx.y * blockDim.y + threadIdx.y >= rowMatrix || blockIdx.x * blockDim.x + threadIdx.x >= colMatrix)
			{
				return;
			}

			shTile[threadIdx.y * BLOCK_SIZE + threadIdx.x] = d_inMatrix[row * colMatrix + col];

			d_out[col * rowMatrix + row] = shTile[threadIdx.y * BLOCK_SIZE + threadIdx.x];
		}

		__global__ void OutPr(float* d_inFirst, float* d_inSecond, float* d_out, int rowFirst, int rowSecond)
		{
			int row = blockIdx.y * blockDim.y + threadIdx.y;
			int col = blockIdx.x * blockDim.x + threadIdx.x;

			if (row >= rowFirst || col >= rowSecond)
			{
				return;
			}

			d_out[row * rowSecond + col] = d_inFirst[row] * d_inSecond[col];
		}

		__global__ void OutPrSH(float* d_inFirst, float* d_inSecond, float* d_out, int rowFirst, int rowSecond)
		{
			int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
			int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

			__shared__ float shFirst[BLOCK_SIZE * BLOCK_SIZE];
			__shared__ float shSecond[BLOCK_SIZE * BLOCK_SIZE];

			float prod = 0.0f;

			if (blockIdx.y * blockDim.y + threadIdx.y >= rowFirst || blockIdx.x * blockDim.x + threadIdx.x >= rowSecond)
			{
				return;
			}

			shFirst[row] = d_inFirst[blockIdx.y * blockDim.y + threadIdx.y];
			shSecond[col] = d_inSecond[blockIdx.x * blockDim.x + threadIdx.x];

			__syncthreads();

			prod = shFirst[row] * shSecond[col];

			d_out[(blockIdx.y * blockDim.y + threadIdx.y) * rowSecond + (blockIdx.x * blockDim.x + threadIdx.x)] = prod;
		}

		__global__ void VecDiff(float* d_inFirst, float* d_inSecond, float* d_out, int rows)
		{
			int row = blockIdx.x * blockDim.x + threadIdx.x;

			if (row >= rows)
			{
				return;
			}

			d_out[row] = d_inFirst[row] - d_inSecond[row];
		}

		__global__ void VecDiffSH(float* d_inFirst, float* d_inSecond, float* d_out, int rows)
		{
			int row = blockIdx.x * blockDim.x + threadIdx.x;

			__shared__ float shFirst[BLOCK_SIZE * BLOCK_SIZE];
			__shared__ float shSecond[BLOCK_SIZE * BLOCK_SIZE];

			float diff = 0.0f;

			if (blockIdx.x * blockDim.x + threadIdx.x >= rows)
			{
				return;
			}

			shFirst[blockIdx.x * BLOCK_SIZE + threadIdx.x] = d_inFirst[blockIdx.x * blockDim.x + threadIdx.x];
			shSecond[blockIdx.x * BLOCK_SIZE + threadIdx.x] = d_inSecond[blockIdx.x * blockDim.x + threadIdx.x];

			__syncthreads();

			diff = shFirst[blockIdx.x * BLOCK_SIZE + threadIdx.x] - shSecond[blockIdx.x * BLOCK_SIZE + threadIdx.x];

			d_out[row] = diff;
		}

		//Functions.
		void matricesDotProductGPU(float* h_first, const int& rowFirst, const int& colFirst, float* h_second, const int& rowSecond, const int& colSecond, float* h_result)
		{
			float* d_inFirst;
			float* d_inSecond;
			float* d_out;

			const int BYFIRST = (rowFirst * colFirst) * sizeof(float);
			const int BYSECOND = (rowSecond * colSecond) * sizeof(float);
			const int BYRESULT = (rowFirst * colSecond) * sizeof(float);

			hipMalloc((void**)&d_inFirst, BYFIRST);
			hipMalloc((void**)&d_inSecond, BYSECOND);
			hipMalloc((void**)&d_out, BYRESULT);

			hipMemcpy(d_inFirst, h_first, BYFIRST, hipMemcpyHostToDevice);
			hipMemcpy(d_inSecond, h_second, BYSECOND, hipMemcpyHostToDevice);

			dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
			dim3 dimGrid((colSecond + dimBlock.x - 1) / dimBlock.x, (rowFirst + dimBlock.y - 1) / dimBlock.y);

			MultipMatrices <<<dimGrid, dimBlock>>>(d_inFirst, d_inSecond, d_out, rowFirst, colFirst, colSecond);

			hipMemcpy(h_result, d_out, BYRESULT, hipMemcpyDeviceToHost);

			hipFree(d_inFirst);
			hipFree(d_inSecond);
			hipFree(d_out);
		}

		void matricesDotProductGPUSH(float* h_first, const int& rowFirst, const int& colFirst, float* h_second, const int& rowSecond, const int& colSecond, float* h_result)
		{
			float* d_inFirst;
			float* d_inSecond;
			float* d_out;

			const int BYFIRST = (rowFirst * colFirst) * sizeof(float);
			const int BYSECOND = (rowSecond * colSecond) * sizeof(float);
			const int BYRESULT = (rowFirst * colSecond) * sizeof(float);

			hipMalloc((void**)&d_inFirst, BYFIRST);
			hipMalloc((void**)&d_inSecond, BYSECOND);
			hipMalloc((void**)&d_out, BYRESULT);

			hipMemcpy(d_inFirst, h_first, BYFIRST, hipMemcpyHostToDevice);
			hipMemcpy(d_inSecond, h_second, BYSECOND, hipMemcpyHostToDevice);

			dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
			dim3 dimGrid((colSecond + dimBlock.x - 1) / dimBlock.x, (rowFirst + dimBlock.y - 1) / dimBlock.y);

			MultipMatricesSH <<<dimGrid, dimBlock>>>(d_inFirst, d_inSecond, d_out, rowFirst, colFirst, colSecond);

			hipMemcpy(h_result, d_out, BYRESULT, hipMemcpyDeviceToHost);

			hipFree(d_inFirst);
			hipFree(d_inSecond);
			hipFree(d_out);
		}

		void transposeGPU(float* h_matrix, const int& rowMatrix, const int& colMatrix, float* h_result)
		{
			float* d_inMatrix;
			float* d_out;

			const int BYTES = (rowMatrix * colMatrix) * sizeof(float);

			hipMalloc((void**)&d_inMatrix, BYTES);
			hipMalloc((void**)&d_out, BYTES);

			hipMemcpy(d_inMatrix, h_matrix, BYTES, hipMemcpyHostToDevice);

			dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
			dim3 dimGrid((colMatrix + dimBlock.x - 1) / dimBlock.x, (rowMatrix + dimBlock.y - 1) / dimBlock.y);

			Transp <<<dimGrid, dimBlock>>>(d_inMatrix, d_out, rowMatrix, colMatrix);

			hipMemcpy(h_result, d_out, BYTES, hipMemcpyDeviceToHost);

			hipFree(d_inMatrix);
			hipFree(d_out);
		}

		void transposeGPUSH(float* h_matrix, const int& rowMatrix, const int& colMatrix, float* h_result)
		{
			float* d_inMatrix;
			float* d_out;

			const int BYTES = (rowMatrix * colMatrix) * sizeof(float);

			hipMalloc((void**)&d_inMatrix, BYTES);
			hipMalloc((void**)&d_out, BYTES);

			hipMemcpy(d_inMatrix, h_matrix, BYTES, hipMemcpyHostToDevice);

			dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
			dim3 dimGrid((colMatrix + dimBlock.x - 1) / dimBlock.x, (rowMatrix + dimBlock.y - 1) / dimBlock.y);

			TranspSH <<<dimGrid, dimBlock>>>(d_inMatrix, d_out, rowMatrix, colMatrix);

			hipMemcpy(h_result, d_out, BYTES, hipMemcpyDeviceToHost);

			hipFree(d_inMatrix);
			hipFree(d_out);
		}

		void outerProdGPU(float* h_first, const int& rowFirst, float* h_second, const int& rowSecond, float* h_result)
		{
			float* d_inFirst;
			float* d_inSecond;
			float* d_out;

			const int BYFIRST = rowFirst * sizeof(float);
			const int BYSECOND = rowSecond * sizeof(float);
			const int BYRESULT = (rowFirst * rowSecond) * sizeof(float);

			hipMalloc((void**)&d_inFirst, BYFIRST);
			hipMalloc((void**)&d_inSecond, BYSECOND);
			hipMalloc((void**)&d_out, BYRESULT);

			hipMemcpy(d_inFirst, h_first, BYFIRST, hipMemcpyHostToDevice);
			hipMemcpy(d_inSecond, h_second, BYSECOND, hipMemcpyHostToDevice);

			dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
			dim3 dimGrid((rowSecond + dimBlock.x - 1) / dimBlock.x, (rowFirst + dimBlock.y - 1) / dimBlock.y);

			OutPr <<<dimGrid, dimBlock>>>(d_inFirst, d_inSecond, d_out, rowFirst, rowSecond);

			hipMemcpy(h_result, d_out, BYRESULT, hipMemcpyDeviceToHost);

			hipFree(d_inFirst);
			hipFree(d_inSecond);
			hipFree(d_out);
		}

		void outerProdGPUSH(float* h_first, const int& rowFirst, float* h_second, const int& rowSecond, float* h_result)
		{
			float* d_inFirst;
			float* d_inSecond;
			float* d_out;

			const int BYFIRST = rowFirst * sizeof(float);
			const int BYSECOND = rowSecond * sizeof(float);
			const int BYRESULT = (rowFirst * rowSecond) * sizeof(float);

			hipMalloc((void**)&d_inFirst, BYFIRST);
			hipMalloc((void**)&d_inSecond, BYSECOND);
			hipMalloc((void**)&d_out, BYRESULT);

			hipMemcpy(d_inFirst, h_first, BYFIRST, hipMemcpyHostToDevice);
			hipMemcpy(d_inSecond, h_second, BYSECOND, hipMemcpyHostToDevice);

			dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
			dim3 dimGrid((rowSecond + dimBlock.x - 1) / dimBlock.x, (rowFirst + dimBlock.y - 1) / dimBlock.y);

			OutPrSH <<<dimGrid, dimBlock>>>(d_inFirst, d_inSecond, d_out, rowFirst, rowSecond);

			hipMemcpy(h_result, d_out, BYRESULT, hipMemcpyDeviceToHost);

			hipFree(d_inFirst);
			hipFree(d_inSecond);
			hipFree(d_out);
		}

		void vectorsDiffGPU(float* h_first, float* h_second, const int& rows, float* h_result)
		{
			float* d_inFirst;
			float* d_inSecond;
			float* d_out;

			const int BYFIRST = rows * sizeof(float);
			const int BYSECOND = rows * sizeof(float);
			const int BYRESULT = rows * sizeof(float);

			hipMalloc((void**)&d_inFirst, BYFIRST);
			hipMalloc((void**)&d_inSecond, BYSECOND);
			hipMalloc((void**)&d_out, BYRESULT);

			hipMemcpy(d_inFirst, h_first, BYFIRST, hipMemcpyHostToDevice);
			hipMemcpy(d_inSecond, h_second, BYSECOND, hipMemcpyHostToDevice);

			dim3 dimBlock(BLOCK_SIZE);
			dim3 dimGrid((rows + dimBlock.x - 1) / dimBlock.x);

			VecDiff <<<dimGrid, dimBlock>>>(d_inFirst, d_inSecond, d_out, rows);

			hipMemcpy(h_result, d_out, BYRESULT, hipMemcpyDeviceToHost);

			hipFree(d_inFirst);
			hipFree(d_inSecond);
			hipFree(d_out);
		}

		void vectorsDiffGPUSH(float* h_first, float* h_second, const int& rows, float* h_result)
		{
			float* d_inFirst;
			float* d_inSecond;
			float* d_out;

			const int BYFIRST = rows * sizeof(float);
			const int BYSECOND = rows * sizeof(float);
			const int BYRESULT = rows * sizeof(float);

			hipMalloc((void**)&d_inFirst, BYFIRST);
			hipMalloc((void**)&d_inSecond, BYSECOND);
			hipMalloc((void**)&d_out, BYRESULT);

			hipMemcpy(d_inFirst, h_first, BYFIRST, hipMemcpyHostToDevice);
			hipMemcpy(d_inSecond, h_second, BYSECOND, hipMemcpyHostToDevice);

			dim3 dimBlock(BLOCK_SIZE);
			dim3 dimGrid((rows + dimBlock.x - 1) / dimBlock.x);

			VecDiffSH <<<dimGrid, dimBlock>>>(d_inFirst, d_inSecond, d_out, rows);

			hipMemcpy(h_result, d_out, BYRESULT, hipMemcpyDeviceToHost);

			hipFree(d_inFirst);
			hipFree(d_inSecond);
			hipFree(d_out);
		}
	}
}





