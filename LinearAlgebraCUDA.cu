#include "hip/hip_runtime.h"
#include "linearAlgebraCUDA.cuh"

namespace WhydahGally
{
	namespace Maths
	{
		//Kernels.
		template<typename T> __global__ void MultipMatrices(T* d_inFirst, T* d_inSecond, T* d_out, int rowFirst, int colFirst, int colSecond)
		{
			int row = blockIdx.y * blockDim.y + threadIdx.y;
			int col = blockIdx.x * blockDim.x + threadIdx.x;

			T sum = 0.0f;

			if (row >= rowFirst || col >= colSecond)
			{
				return;
			}

			for (int i = 0; i < colFirst; i++)
			{
				sum += d_inFirst[row * colFirst + i] * d_inSecond[i * colSecond + col];
			}

			d_out[row * colSecond + col] = sum;
		}

		template<typename T> __global__ void MultipMatricesSH(T* d_inFirst, T* d_inSecond, T* d_out, int rowFirst, int colFirst, int colSecond)
		{
			int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
			int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

			__shared__ T shFirst[BLOCK_SIZE * BLOCK_SIZE];
			__shared__ T shSecond[BLOCK_SIZE * BLOCK_SIZE];

			T sum = 0.0f;

			for (int i = 0; i < (BLOCK_SIZE + colFirst - 1) / BLOCK_SIZE; i++)
			{
				if (i * BLOCK_SIZE + threadIdx.x < colFirst && row < rowFirst)
				{
					shFirst[threadIdx.y * BLOCK_SIZE + threadIdx.x] = d_inFirst[row * colFirst + i * BLOCK_SIZE + threadIdx.x];
				}
				else
				{
					shFirst[threadIdx.y * BLOCK_SIZE + threadIdx.x] = 0.0f;
				}

				if (i * BLOCK_SIZE + threadIdx.y < colFirst && col < colSecond)
				{
					shSecond[threadIdx.y * BLOCK_SIZE + threadIdx.x] = d_inSecond[(i * BLOCK_SIZE + threadIdx.y) * colSecond + col];
				}
				else
				{
					shSecond[threadIdx.y * BLOCK_SIZE + threadIdx.x] = 0.0f;
				}

				__syncthreads();

				for (int j = 0; j < BLOCK_SIZE; j++)
				{
					sum += shFirst[threadIdx.y * BLOCK_SIZE + j] * shSecond[j * BLOCK_SIZE + threadIdx.x];
				}

				__syncthreads();
			}

			if (row < rowFirst && col < colSecond)
			{
				d_out[((blockIdx.y * blockDim.y + threadIdx.y) * colSecond) + (blockIdx.x * blockDim.x) + threadIdx.x] = sum;
			}
		}

		template<typename T> __global__ void Transp(T* d_inMatrix, T* d_out, int rowMatrix, int colMatrix)
		{
			int row = blockIdx.y * blockDim.y + threadIdx.y;
			int col = blockIdx.x * blockDim.x + threadIdx.x;

			if (row >= rowMatrix || col >= colMatrix)
			{
				return;
			}

			d_out[col * rowMatrix + row] = d_inMatrix[row * colMatrix + col];
		}

		template<typename T> __global__ void TranspSH(T *d_inMatrix, T *d_out, int rowMatrix, int colMatrix)
		{
			int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
			int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

			__shared__ T shTile[BLOCK_SIZE * BLOCK_SIZE];

			if (blockIdx.y * blockDim.y + threadIdx.y >= rowMatrix || blockIdx.x * blockDim.x + threadIdx.x >= colMatrix)
			{
				return;
			}

			shTile[threadIdx.y * BLOCK_SIZE + threadIdx.x] = d_inMatrix[row * colMatrix + col];

			__syncthreads();

			d_out[col * rowMatrix + row] = shTile[threadIdx.y * BLOCK_SIZE + threadIdx.x];
		}

		template<typename T> __global__ void OutPr(T* d_inFirst, T* d_inSecond, T* d_out, int rowFirst, int rowSecond)
		{
			int row = blockIdx.y * blockDim.y + threadIdx.y;
			int col = blockIdx.x * blockDim.x + threadIdx.x;

			if (row >= rowFirst || col >= rowSecond)
			{
				return;
			}

			d_out[row * rowSecond + col] = d_inFirst[row] * d_inSecond[col];
		}

		template<typename T> __global__ void OutPrSH(T* d_inFirst, T* d_inSecond, T* d_out, int rowFirst, int rowSecond)
		{
			int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
			int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

			__shared__ T shFirst[BLOCK_SIZE * BLOCK_SIZE];
			__shared__ T shSecond[BLOCK_SIZE * BLOCK_SIZE];

			T prod = 0.0f;

			if (blockIdx.y * blockDim.y + threadIdx.y >= rowFirst || blockIdx.x * blockDim.x + threadIdx.x >= rowSecond)
			{
				return;
			}

			shFirst[row] = d_inFirst[blockIdx.y * blockDim.y + threadIdx.y];
			shSecond[col] = d_inSecond[blockIdx.x * blockDim.x + threadIdx.x];

			__syncthreads();

			prod = shFirst[row] * shSecond[col];

			d_out[(blockIdx.y * blockDim.y + threadIdx.y) * rowSecond + (blockIdx.x * blockDim.x + threadIdx.x)] = prod;
		}

		template<typename T> __global__ void VecDiff(T* d_inFirst, T* d_inSecond, T* d_out, int rows)
		{
			int row = blockIdx.x * blockDim.x + threadIdx.x;

			if (row >= rows)
			{
				return;
			}

			d_out[row] = d_inFirst[row] - d_inSecond[row];
		}

		template<typename T> __global__ void VecDiffSH(T* d_inFirst, T* d_inSecond, T* d_out, int rows)
		{
			int row = blockIdx.x * blockDim.x + threadIdx.x;

			__shared__ T shFirst[BLOCK_SIZE * BLOCK_SIZE];
			__shared__ T shSecond[BLOCK_SIZE * BLOCK_SIZE];

			T diff = 0.0f;

			if (row >= rows)
			{
				return;
			}

			shFirst[blockIdx.x * BLOCK_SIZE + threadIdx.x] = d_inFirst[row];
			shSecond[blockIdx.x * BLOCK_SIZE + threadIdx.x] = d_inSecond[row];

			__syncthreads();

			diff = shFirst[blockIdx.x * BLOCK_SIZE + threadIdx.x] - shSecond[blockIdx.x * BLOCK_SIZE + threadIdx.x];

			d_out[row] = diff;
		}

		//Functions.
		template<typename T> void matricesDotProductGPU(T* h_first, const int& rowFirst, const int& colFirst, T* h_second, const int& rowSecond, const int& colSecond, T* h_result)
		{
			T* d_inFirst;
			T* d_inSecond;
			T* d_out;

			const int BYFIRST = (rowFirst * colFirst) * sizeof(T);
			const int BYSECOND = (rowSecond * colSecond) * sizeof(T);
			const int BYRESULT = (rowFirst * colSecond) * sizeof(T);

			hipMalloc((void**)&d_inFirst, BYFIRST);
			hipMalloc((void**)&d_inSecond, BYSECOND);
			hipMalloc((void**)&d_out, BYRESULT);

			hipMemcpy(d_inFirst, h_first, BYFIRST, hipMemcpyHostToDevice);
			hipMemcpy(d_inSecond, h_second, BYSECOND, hipMemcpyHostToDevice);

			dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
			dim3 dimGrid((colSecond + dimBlock.x - 1) / dimBlock.x, (rowFirst + dimBlock.y - 1) / dimBlock.y);

			MultipMatrices <<<dimGrid, dimBlock>>>(d_inFirst, d_inSecond, d_out, rowFirst, colFirst, colSecond);

			hipMemcpy(h_result, d_out, BYRESULT, hipMemcpyDeviceToHost);

			hipFree(d_inFirst);
			hipFree(d_inSecond);
			hipFree(d_out);
		}

		template<typename T> void matricesDotProductGPUSH(T* h_first, const int& rowFirst, const int& colFirst, T* h_second, const int& rowSecond, const int& colSecond, T* h_result)
		{
			T* d_inFirst;
			T* d_inSecond;
			T* d_out;

			const int BYFIRST = (rowFirst * colFirst) * sizeof(T);
			const int BYSECOND = (rowSecond * colSecond) * sizeof(T);
			const int BYRESULT = (rowFirst * colSecond) * sizeof(T);

			hipMalloc((void**)&d_inFirst, BYFIRST);
			hipMalloc((void**)&d_inSecond, BYSECOND);
			hipMalloc((void**)&d_out, BYRESULT);

			hipMemcpy(d_inFirst, h_first, BYFIRST, hipMemcpyHostToDevice);
			hipMemcpy(d_inSecond, h_second, BYSECOND, hipMemcpyHostToDevice);

			dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
			dim3 dimGrid((colSecond + dimBlock.x - 1) / dimBlock.x, (rowFirst + dimBlock.y - 1) / dimBlock.y);

			MultipMatricesSH <<<dimGrid, dimBlock>>>(d_inFirst, d_inSecond, d_out, rowFirst, colFirst, colSecond);

			hipMemcpy(h_result, d_out, BYRESULT, hipMemcpyDeviceToHost);

			hipFree(d_inFirst);
			hipFree(d_inSecond);
			hipFree(d_out);
		}

		template<typename T> void transposeGPU(T* h_matrix, const int& rowMatrix, const int& colMatrix, T* h_result)
		{
			T* d_inMatrix;
			T* d_out;

			const int BYTES = (rowMatrix * colMatrix) * sizeof(T);

			hipMalloc((void**)&d_inMatrix, BYTES);
			hipMalloc((void**)&d_out, BYTES);

			hipMemcpy(d_inMatrix, h_matrix, BYTES, hipMemcpyHostToDevice);

			dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
			dim3 dimGrid((colMatrix + dimBlock.x - 1) / dimBlock.x, (rowMatrix + dimBlock.y - 1) / dimBlock.y);

			Transp <<<dimGrid, dimBlock>>>(d_inMatrix, d_out, rowMatrix, colMatrix);

			hipMemcpy(h_result, d_out, BYTES, hipMemcpyDeviceToHost);

			hipFree(d_inMatrix);
			hipFree(d_out);
		}

		template<typename T> void transposeGPUSH(T* h_matrix, const int& rowMatrix, const int& colMatrix, T* h_result)
		{
			T* d_inMatrix;
			T* d_out;

			const int BYTES = (rowMatrix * colMatrix) * sizeof(T);

			hipMalloc((void**)&d_inMatrix, BYTES);
			hipMalloc((void**)&d_out, BYTES);

			hipMemcpy(d_inMatrix, h_matrix, BYTES, hipMemcpyHostToDevice);

			dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
			dim3 dimGrid((colMatrix + dimBlock.x - 1) / dimBlock.x, (rowMatrix + dimBlock.y - 1) / dimBlock.y);

			TranspSH <<<dimGrid, dimBlock>>>(d_inMatrix, d_out, rowMatrix, colMatrix);

			hipMemcpy(h_result, d_out, BYTES, hipMemcpyDeviceToHost);

			hipFree(d_inMatrix);
			hipFree(d_out);
		}

		template<typename T> void outerProdGPU(T* h_first, const int& rowFirst, T* h_second, const int& rowSecond, T* h_result)
		{
			T* d_inFirst;
			T* d_inSecond;
			T* d_out;

			const int BYFIRST = rowFirst * sizeof(T);
			const int BYSECOND = rowSecond * sizeof(T);
			const int BYRESULT = (rowFirst * rowSecond) * sizeof(T);

			hipMalloc((void**)&d_inFirst, BYFIRST);
			hipMalloc((void**)&d_inSecond, BYSECOND);
			hipMalloc((void**)&d_out, BYRESULT);

			hipMemcpy(d_inFirst, h_first, BYFIRST, hipMemcpyHostToDevice);
			hipMemcpy(d_inSecond, h_second, BYSECOND, hipMemcpyHostToDevice);

			dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
			dim3 dimGrid((rowSecond + dimBlock.x - 1) / dimBlock.x, (rowFirst + dimBlock.y - 1) / dimBlock.y);

			OutPr <<<dimGrid, dimBlock>>>(d_inFirst, d_inSecond, d_out, rowFirst, rowSecond);

			hipMemcpy(h_result, d_out, BYRESULT, hipMemcpyDeviceToHost);

			hipFree(d_inFirst);
			hipFree(d_inSecond);
			hipFree(d_out);
		}

		template<typename T> void outerProdGPUSH(T* h_first, const int& rowFirst, T* h_second, const int& rowSecond, T* h_result)
		{
			T* d_inFirst;
			T* d_inSecond;
			T* d_out;

			const int BYFIRST = rowFirst * sizeof(T);
			const int BYSECOND = rowSecond * sizeof(T);
			const int BYRESULT = (rowFirst * rowSecond) * sizeof(T);

			hipMalloc((void**)&d_inFirst, BYFIRST);
			hipMalloc((void**)&d_inSecond, BYSECOND);
			hipMalloc((void**)&d_out, BYRESULT);

			hipMemcpy(d_inFirst, h_first, BYFIRST, hipMemcpyHostToDevice);
			hipMemcpy(d_inSecond, h_second, BYSECOND, hipMemcpyHostToDevice);

			dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
			dim3 dimGrid((rowSecond + dimBlock.x - 1) / dimBlock.x, (rowFirst + dimBlock.y - 1) / dimBlock.y);

			OutPrSH <<<dimGrid, dimBlock>>>(d_inFirst, d_inSecond, d_out, rowFirst, rowSecond);

			hipMemcpy(h_result, d_out, BYRESULT, hipMemcpyDeviceToHost);

			hipFree(d_inFirst);
			hipFree(d_inSecond);
			hipFree(d_out);
		}

		template<typename T> void vectorsDiffGPU(T* h_first, T* h_second, const int& rows, T* h_result)
		{
			T* d_inFirst;
			T* d_inSecond;
			T* d_out;

			const int BYFIRST = rows * sizeof(T);
			const int BYSECOND = rows * sizeof(T);
			const int BYRESULT = rows * sizeof(T);

			hipMalloc((void**)&d_inFirst, BYFIRST);
			hipMalloc((void**)&d_inSecond, BYSECOND);
			hipMalloc((void**)&d_out, BYRESULT);

			hipMemcpy(d_inFirst, h_first, BYFIRST, hipMemcpyHostToDevice);
			hipMemcpy(d_inSecond, h_second, BYSECOND, hipMemcpyHostToDevice);

			dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
			dim3 dimGrid((rows + dimBlock.x - 1) / dimBlock.x);

			VecDiff <<<dimGrid, dimBlock>>>(d_inFirst, d_inSecond, d_out, rows);

			hipMemcpy(h_result, d_out, BYRESULT, hipMemcpyDeviceToHost);

			hipFree(d_inFirst);
			hipFree(d_inSecond);
			hipFree(d_out);
		}

		template<typename T> void vectorsDiffGPUSH(T* h_first, T* h_second, const int& rows, T* h_result)
		{
			T* d_inFirst;
			T* d_inSecond;
			T* d_out;

			const int BYFIRST = rows * sizeof(T);
			const int BYSECOND = rows * sizeof(T);
			const int BYRESULT = rows * sizeof(T);

			hipMalloc((void**)&d_inFirst, BYFIRST);
			hipMalloc((void**)&d_inSecond, BYSECOND);
			hipMalloc((void**)&d_out, BYRESULT);

			hipMemcpy(d_inFirst, h_first, BYFIRST, hipMemcpyHostToDevice);
			hipMemcpy(d_inSecond, h_second, BYSECOND, hipMemcpyHostToDevice);

			dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
			dim3 dimGrid((rows + dimBlock.x - 1) / dimBlock.x);

			VecDiffSH <<<dimGrid, dimBlock>>>(d_inFirst, d_inSecond, d_out, rows);

			hipMemcpy(h_result, d_out, BYRESULT, hipMemcpyDeviceToHost);

			hipFree(d_inFirst);
			hipFree(d_inSecond);
			hipFree(d_out);
		}

		//Forced template instantiations.
		//float
		template __global__ void MultipMatrices(float* d_inFirst, float* d_inSecond, float* d_out, int rowFirst, int colFirst, int colSecond);
		template __global__ void MultipMatricesSH(float* d_inFirst, float* d_inSecond, float* d_out, int rowFirst, int colFirst, int colSecond);

		template __global__ void Transp(float* d_inMatrix, float* d_out, int rowMatrix, int colMatrix);
		template __global__ void TranspSH(float *d_inMatrix, float *d_out, int rowMatrix, int colMatrix);

		template __global__ void OutPr(float* d_inFirst, float* d_inSecond, float* d_out, int rowFirst, int rowSecond);
		template __global__ void OutPrSH(float* d_inFirst, float* d_inSecond, float* d_out, int rowFirst, int rowSecond);

		template __global__ void VecDiff(float* d_inFirst, float* d_inSecond, float* d_out, int rows);
		template __global__ void VecDiffSH(float* d_inFirst, float* d_inSecond, float* d_out, int rows);

		template void matricesDotProductGPU(float* h_first, const int& rowFirst, const int& colFirst, float* h_second, const int& rowSecond, const int& colSecond, float* h_result);
		template void matricesDotProductGPUSH(float* h_first, const int& rowFirst, const int& colFirst, float* h_second, const int& rowSecond, const int& colSecond, float* h_result);

		template void transposeGPU(float* h_matrix, const int& rowMatrix, const int& colMatrix, float* h_result);
		template void transposeGPUSH(float* h_matrix, const int& rowMatrix, const int& colMatrix, float* h_result);

		template void outerProdGPU(float* h_first, const int& rowFirst, float* h_second, const int& rowSecond, float* h_result);
		template void outerProdGPUSH(float* h_first, const int& rowFirst, float* h_second, const int& rowSecond, float* h_result);

		template void vectorsDiffGPU(float* h_first, float* h_second, const int& rows, float* h_result);
		template void vectorsDiffGPUSH(float* h_first, float* h_second, const int& rows, float* h_result);

		//double
		template __global__ void MultipMatrices(double* d_inFirst, double* d_inSecond, double* d_out, int rowFirst, int colFirst, int colSecond);
		template __global__ void MultipMatricesSH(double* d_inFirst, double* d_inSecond, double* d_out, int rowFirst, int colFirst, int colSecond);

		template __global__ void Transp(double* d_inMatrix, double* d_out, int rowMatrix, int colMatrix);
		template __global__ void TranspSH(double *d_inMatrix, double *d_out, int rowMatrix, int colMatrix);

		template __global__ void OutPr(double* d_inFirst, double* d_inSecond, double* d_out, int rowFirst, int rowSecond);
		template __global__ void OutPrSH(double* d_inFirst, double* d_inSecond, double* d_out, int rowFirst, int rowSecond);

		template __global__ void VecDiff(double* d_inFirst, double* d_inSecond, double* d_out, int rows);
		template __global__ void VecDiffSH(double* d_inFirst, double* d_inSecond, double* d_out, int rows);

		template void matricesDotProductGPU(double* h_first, const int& rowFirst, const int& colFirst, double* h_second, const int& rowSecond, const int& colSecond, double* h_result);
		template void matricesDotProductGPUSH(double* h_first, const int& rowFirst, const int& colFirst, double* h_second, const int& rowSecond, const int& colSecond, double* h_result);

		template void transposeGPU(double* h_matrix, const int& rowMatrix, const int& colMatrix, double* h_result);
		template void transposeGPUSH(double* h_matrix, const int& rowMatrix, const int& colMatrix, double* h_result);

		template void outerProdGPU(double* h_first, const int& rowFirst, double* h_second, const int& rowSecond, double* h_result);
		template void outerProdGPUSH(double* h_first, const int& rowFirst, double* h_second, const int& rowSecond, double* h_result);

		template void vectorsDiffGPU(double* h_first, double* h_second, const int& rows, double* h_result);
		template void vectorsDiffGPUSH(double* h_first, double* h_second, const int& rows, double* h_result);

		//int
		template __global__ void MultipMatrices(int* d_inFirst, int* d_inSecond, int* d_out, int rowFirst, int colFirst, int colSecond);
		template __global__ void MultipMatricesSH(int* d_inFirst, int* d_inSecond, int* d_out, int rowFirst, int colFirst, int colSecond);

		template __global__ void Transp(int* d_inMatrix, int* d_out, int rowMatrix, int colMatrix);
		template __global__ void TranspSH(int *d_inMatrix, int *d_out, int rowMatrix, int colMatrix);

		template __global__ void OutPr(int* d_inFirst, int* d_inSecond, int* d_out, int rowFirst, int rowSecond);
		template __global__ void OutPrSH(int* d_inFirst, int* d_inSecond, int* d_out, int rowFirst, int rowSecond);

		template __global__ void VecDiff(int* d_inFirst, int* d_inSecond, int* d_out, int rows);
		template __global__ void VecDiffSH(int* d_inFirst, int* d_inSecond, int* d_out, int rows);

		template void matricesDotProductGPU(int* h_first, const int& rowFirst, const int& colFirst, int* h_second, const int& rowSecond, const int& colSecond, int* h_result);
		template void matricesDotProductGPUSH(int* h_first, const int& rowFirst, const int& colFirst, int* h_second, const int& rowSecond, const int& colSecond, int* h_result);

		template void transposeGPU(int* h_matrix, const int& rowMatrix, const int& colMatrix, int* h_result);
		template void transposeGPUSH(int* h_matrix, const int& rowMatrix, const int& colMatrix, int* h_result);

		template void outerProdGPU(int* h_first, const int& rowFirst, int* h_second, const int& rowSecond, int* h_result);
		template void outerProdGPUSH(int* h_first, const int& rowFirst, int* h_second, const int& rowSecond, int* h_result);

		template void vectorsDiffGPU(int* h_first, int* h_second, const int& rows, int* h_result);
		template void vectorsDiffGPUSH(int* h_first, int* h_second, const int& rows, int* h_result);

		//unsigned int
		template __global__ void MultipMatrices(unsigned int* d_inFirst, unsigned int* d_inSecond, unsigned int* d_out, int rowFirst, int colFirst, int colSecond);
		template __global__ void MultipMatricesSH(unsigned int* d_inFirst, unsigned int* d_inSecond, unsigned int* d_out, int rowFirst, int colFirst, int colSecond);

		template __global__ void Transp(unsigned int* d_inMatrix, unsigned int* d_out, int rowMatrix, int colMatrix);
		template __global__ void TranspSH(unsigned int *d_inMatrix, unsigned int *d_out, int rowMatrix, int colMatrix);

		template __global__ void OutPr(unsigned int* d_inFirst, unsigned int* d_inSecond, unsigned int* d_out, int rowFirst, int rowSecond);
		template __global__ void OutPrSH(unsigned int* d_inFirst, unsigned int* d_inSecond, unsigned int* d_out, int rowFirst, int rowSecond);

		template __global__ void VecDiff(unsigned int* d_inFirst, unsigned int* d_inSecond, unsigned int* d_out, int rows);
		template __global__ void VecDiffSH(unsigned int* d_inFirst, unsigned int* d_inSecond, unsigned int* d_out, int rows);

		template void matricesDotProductGPU(unsigned int* h_first, const int& rowFirst, const int& colFirst, unsigned int* h_second, const int& rowSecond, const int& colSecond, unsigned int* h_result);
		template void matricesDotProductGPUSH(unsigned int* h_first, const int& rowFirst, const int& colFirst, unsigned int* h_second, const int& rowSecond, const int& colSecond, unsigned int* h_result);

		template void transposeGPU(unsigned int* h_matrix, const int& rowMatrix, const int& colMatrix, unsigned int* h_result);
		template void transposeGPUSH(unsigned int* h_matrix, const int& rowMatrix, const int& colMatrix, unsigned int* h_result);

		template void outerProdGPU(unsigned int* h_first, const int& rowFirst, unsigned int* h_second, const int& rowSecond, unsigned int* h_result);
		template void outerProdGPUSH(unsigned int* h_first, const int& rowFirst, unsigned int* h_second, const int& rowSecond, unsigned int* h_result);

		template void vectorsDiffGPU(unsigned int* h_first, unsigned int* h_second, const int& rows, unsigned int* h_result);
		template void vectorsDiffGPUSH(unsigned int* h_first, unsigned int* h_second, const int& rows, unsigned int* h_result);

		//long
		template __global__ void MultipMatrices(long* d_inFirst, long* d_inSecond, long* d_out, int rowFirst, int colFirst, int colSecond);
		template __global__ void MultipMatricesSH(long* d_inFirst, long* d_inSecond, long* d_out, int rowFirst, int colFirst, int colSecond);

		template __global__ void Transp(long* d_inMatrix, long* d_out, int rowMatrix, int colMatrix);
		template __global__ void TranspSH(long *d_inMatrix, long *d_out, int rowMatrix, int colMatrix);

		template __global__ void OutPr(long* d_inFirst, long* d_inSecond, long* d_out, int rowFirst, int rowSecond);
		template __global__ void OutPrSH(long* d_inFirst, long* d_inSecond, long* d_out, int rowFirst, int rowSecond);

		template __global__ void VecDiff(long* d_inFirst, long* d_inSecond, long* d_out, int rows);
		template __global__ void VecDiffSH(long* d_inFirst, long* d_inSecond, long* d_out, int rows);

		template void matricesDotProductGPU(long* h_first, const int& rowFirst, const int& colFirst, long* h_second, const int& rowSecond, const int& colSecond, long* h_result);
		template void matricesDotProductGPUSH(long* h_first, const int& rowFirst, const int& colFirst, long* h_second, const int& rowSecond, const int& colSecond, long* h_result);

		template void transposeGPU(long* h_matrix, const int& rowMatrix, const int& colMatrix, long* h_result);
		template void transposeGPUSH(long* h_matrix, const int& rowMatrix, const int& colMatrix, long* h_result);

		template void outerProdGPU(long* h_first, const int& rowFirst, long* h_second, const int& rowSecond, long* h_result);
		template void outerProdGPUSH(long* h_first, const int& rowFirst, long* h_second, const int& rowSecond, long* h_result);

		template void vectorsDiffGPU(long* h_first, long* h_second, const int& rows, long* h_result);
		template void vectorsDiffGPUSH(long* h_first, long* h_second, const int& rows, long* h_result);

		//unsigned long
		template __global__ void MultipMatrices(unsigned long* d_inFirst, unsigned long* d_inSecond, unsigned long* d_out, int rowFirst, int colFirst, int colSecond);
		template __global__ void MultipMatricesSH(unsigned long* d_inFirst, unsigned long* d_inSecond, unsigned long* d_out, int rowFirst, int colFirst, int colSecond);

		template __global__ void Transp(unsigned long* d_inMatrix, unsigned long* d_out, int rowMatrix, int colMatrix);
		template __global__ void TranspSH(unsigned long *d_inMatrix, unsigned long *d_out, int rowMatrix, int colMatrix);

		template __global__ void OutPr(unsigned long* d_inFirst, unsigned long* d_inSecond, unsigned long* d_out, int rowFirst, int rowSecond);
		template __global__ void OutPrSH(unsigned long* d_inFirst, unsigned long* d_inSecond, unsigned long* d_out, int rowFirst, int rowSecond);

		template __global__ void VecDiff(unsigned long* d_inFirst, unsigned long* d_inSecond, unsigned long* d_out, int rows);
		template __global__ void VecDiffSH(unsigned long* d_inFirst, unsigned long* d_inSecond, unsigned long* d_out, int rows);

		template void matricesDotProductGPU(unsigned long* h_first, const int& rowFirst, const int& colFirst, unsigned long* h_second, const int& rowSecond, const int& colSecond, unsigned long* h_result);
		template void matricesDotProductGPUSH(unsigned long* h_first, const int& rowFirst, const int& colFirst, unsigned long* h_second, const int& rowSecond, const int& colSecond, unsigned long* h_result);

		template void transposeGPU(unsigned long* h_matrix, const int& rowMatrix, const int& colMatrix, unsigned long* h_result);
		template void transposeGPUSH(unsigned long* h_matrix, const int& rowMatrix, const int& colMatrix, unsigned long* h_result);

		template void outerProdGPU(unsigned long* h_first, const int& rowFirst, unsigned long* h_second, const int& rowSecond, unsigned long* h_result);
		template void outerProdGPUSH(unsigned long* h_first, const int& rowFirst, unsigned long* h_second, const int& rowSecond, unsigned long* h_result);

		template void vectorsDiffGPU(unsigned long* h_first, unsigned long* h_second, const int& rows, unsigned long* h_result);
		template void vectorsDiffGPUSH(unsigned long* h_first, unsigned long* h_second, const int& rows, unsigned long* h_result);
	}
}





